#include "hip/hip_runtime.h"
#include "rules.h"

__device__ bool hasLiberty(int x, int y, bool black) const {
	if (isEmpty(x, y)) {
		return true;
	} else if (isBlack(x, y) != black)
	  return false;

	if (x != 0) // Checks if not on left edge
		if (hasLiberty(x - 1, y, black))
			return true;

	if (x != dimensions() - 1) // Checks if not on right edge
		if (hasLiberty(x + 1, y, black))
			return true;

	if (y != 0)  // Checks if not on top edge
		if (hasLiberty(x, y - 1, black))
			return true;

	if (y != dimensions() - 1) // Checks if not on bottom edge
		if (hasLiberty(x, y + 1, black))
			return true;

	return false;
}

__device__ int pointWillBeCaptured(unsigned char x, unsigned char y,
		bool isBlack) const {
	if (isEmpty(x, y))
		throw Point::pointoccupiedexception();
	else {
		return not hasLiberty(x, y, isBlack);
	}
}


//__global__ void captureStones(Point** d_points, unsigned char dimension)
__global__ void captureStones(Point** d_points)
{
			if (not isEmpty(x, y) and isBlack(x, y) == black
					and pointWillBeCaptured(x, y, black))
				capture(threadIdx.x, threadIdx.y);
}

bool willBeTakenCuda(unsigned char DIMENSION, Point** h_points,
		const unsigned char x, const unsigned char y, const bool black)
{
	const size_t size = N(DIMENSION) * sizeof(Point*);
	Point **d_points;// = new Point[N(DIMENSION)];

	hipError_t error;

	error = hipMalloc((void**)&d_points, size);

	if (error != hipSuccess) {
		printf("hipMalloc d_points returned error code %d, line(%d)\n", error,
				__LINE__);
		exit (EXIT_FAILURE);
	}

	//	//debug to check if host board's empty
	//	for (short row = 0; row < DIMENSION; ++row)
	//		for (short col = 0; col < DIMENSION; ++col)
	//			printf("%d mIs_empty %d\n", row * DIMENSION + col,
	//					h_points[row * DIMENSION + col]->mIs_empty);

	//host to device
	error = hipMemcpy(d_points, *h_points, size, hipMemcpyHostToDevice);
	//	error = hipMemcpy(d_points, h_points, sizeof(Point*), hipMemcpyHostToDevice);

	if (error != hipSuccess) {
		printf(
				"hipMemcpy (d_points, &h_points) returned error code %d, line(%d)\n",
				error, __LINE__);
		exit (EXIT_FAILURE);
	}

	captureStones<<<1, N(DIMENSION)>>>(d_points);

	//	for (short row = 0; row < DIMENSION; ++row)	{
	//		for (short col = 0; col < DIMENSION; ++col)
	//		{
	//			h_points[row * DIMENSION + col]->mIs_black = true;
	//			h_points[row * DIMENSION + col]->mIs_empty = false;
	//		}
	//	}

	//device to host
	error = hipMemcpy(*h_points, d_points, size, hipMemcpyDeviceToHost);
	//	error = hipMemcpy(&h_points[0], d_points, size, hipMemcpyDeviceToHost);

	if (error != hipSuccess) {
		printf("hipMemcpy (h_points, d_points) returned error code %d, line(%d)\n",
				error, __LINE__);
		exit (EXIT_FAILURE);
	}

	//	error = hipFree(d_points);
	//
	//	if (error != hipSuccess) {
	//		printf("hipFree(d_points) returned error code %d, line(%d)\n",
	//				error, __LINE__);
	//		exit (EXIT_FAILURE);
	//	}

	return false;
}
